#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <array>
#include <assert.h>
#include <iostream>

namespace {

    template <typename T, int N>
    struct Stride {
        T* data;
        // ptrdiff_t* strides;
        // ptrdiff_t* sizes;
        ptrdiff_t strides[N];
        ptrdiff_t sizes[N];

        // static const int bytes = sizeof(ptrdiff_t) * N;

        Stride(const at::Tensor& t) : data(t.data<T>()) {
            static_assert(sizeof(Stride<T, N>) < sizeof(void*) * 16,
                          "CUDA kernel launch will exeed resources");
            // hipMalloc(&strides, bytes);
            // hipMalloc(&sizes, bytes);
            // hipMemcpy(strides, t.strides().data(), bytes, hipMemcpyHostToDevice);
            // hipMemcpy(sizes, t.sizes().data(), bytes, hipMemcpyHostToDevice);

            std::copy(t.strides().begin(), t.strides().end(), strides);
            std::copy(t.sizes().begin(), t.sizes().end(), sizes);
        }

        __device__
        T* pointer(std::initializer_list<ptrdiff_t> il) {
            return const_cast<T*>(static_cast<const Stride<T, N>&>(*this).pointer(il));
        }

        __device__
        const T* pointer(std::initializer_list<ptrdiff_t> il) const {
            ptrdiff_t ret = 0;
            int n = 0;
            for (auto i : il) {
                assert(0 <= i);
                assert(i < this->sizes[n]);
                ret += i * this->strides[n];
                ++n;
            }
            return data + ret;
        }

    };

#define PARALLEL_FOR(i, n) \
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

    template <typename scalar_t>
    __global__ void unfold_dot_cuda_forward_kernel(
        Stride<scalar_t, 4> ret_tensor,         // (batch, head, time1, restrict)
        const Stride<scalar_t, 4> query_tensor, // (batch, head, time1, feat)
        const Stride<scalar_t, 4> key_tensor,   // (batch, head, time2, feat)
        size_t parallel_size
        ) {
        const auto head_size = ret_tensor.sizes[1];
        const auto time_query = query_tensor.sizes[2];
        const auto time_key = key_tensor.sizes[2];
        const auto feat_size = query_tensor.sizes[3];
        const auto restrict_size = ret_tensor.sizes[3];
        const ptrdiff_t window = (restrict_size - 1) / 2;
        // parallel for batch, head, time_query
        PARALLEL_FOR(i, parallel_size) {
            const ptrdiff_t batch_index = i / (head_size * time_query);
            const ptrdiff_t head_index = (i % (head_size * time_query)) / time_query;
            const ptrdiff_t time_query_index = i % time_query;

            auto* ret_i = ret_tensor.pointer({batch_index, head_index, time_query_index, 0});
            const auto* query_i = query_tensor.pointer({batch_index, head_index, time_query_index, 0});
            const auto* key_i = key_tensor.pointer({batch_index, head_index, time_query_index, 0});

            for (ptrdiff_t w = -window; w <= window; ++w) {
                const ptrdiff_t time_key_index = time_query_index + w;
                if (time_key_index < 0) continue;
                if (time_key_index >= time_key) break;

                // TODO parallel reduction
                scalar_t acc = 0;
                for (ptrdiff_t f = 0; f < feat_size; ++f) {
                    acc += query_i[f * query_tensor.strides[3]] * key_i[w * key_tensor.strides[2] + f * key_tensor.strides[3]];
                }
                ret_i[(w + window) * ret_tensor.strides[3]] = acc;
            }
        }
    }


    template <typename scalar_t>
    __global__ void unfold_dot_cuda_backward_kernel(
        Stride<scalar_t, 4> dquery_tensor,      // (batch, head, time1, feat)
        Stride<scalar_t, 4> dkey_tensor,        // (batch, head, time2, feat)
        const Stride<scalar_t, 4> dret_tensor,  // (batch, head, time1, restrict)
        const Stride<scalar_t, 4> query_tensor, // (batch, head, time1, feat)
        const Stride<scalar_t, 4> key_tensor,   // (batch, head, time2, feat)
        size_t parallel_size
        )
    {
        const auto head_size = dret_tensor.sizes[1];
        const auto time_query = query_tensor.sizes[2];
        const auto time_key = key_tensor.sizes[2];
        const auto feat_size = query_tensor.sizes[3];
        const auto restrict_size = dret_tensor.sizes[3];

        const ptrdiff_t window = (restrict_size - 1) / 2;
        const ptrdiff_t rev_offset = restrict_size - 1 - window;

        // parallel for each (batch, head, time1, feat). sequential for each (restrict)
        PARALLEL_FOR(i, parallel_size) {
            const ptrdiff_t batch_head_index = i / (time_query * feat_size);
            const ptrdiff_t remain = i % (time_query * feat_size);
            const ptrdiff_t time_query_index = remain / feat_size;
            const ptrdiff_t feat_index = remain % feat_size;
            const ptrdiff_t batch_index = batch_head_index / head_size;
            const ptrdiff_t head_index = batch_head_index % head_size;

            const auto* dret_i = dret_tensor.pointer({batch_index, head_index, time_query_index, 0});
            const auto* query_i = query_tensor.pointer({batch_index, head_index, time_query_index, feat_index});
            const auto* key_i = key_tensor.pointer({batch_index, head_index, time_query_index, feat_index});
            scalar_t dquery_i = 0;
            scalar_t dkey_i = 0;

            // for (auto w = -min(time_query_index, window); w <= min(time_query - time_query_index + 1, window); ++w) {
            for (auto w = -window; w <= window; ++w) {
                auto t = time_query_index + w;
                if (0 <= t && t < time_key) {
                    dquery_i += dret_i[(w + window) * dret_tensor.strides[3]] * key_i[w * key_tensor.strides[2]];
                    dkey_i += dret_i[w * dret_tensor.strides[2] + (rev_offset - w) * dret_tensor.strides[3]]
                        * query_i[w * query_tensor.strides[2]];
                }
            }
            *dquery_tensor.pointer({batch_index, head_index, time_query_index, feat_index}) = dquery_i;
            *dkey_tensor.pointer({batch_index, head_index, time_query_index, feat_index}) = dkey_i;
        }
    }

    template <typename scalar_t>
    __global__ void unfold_matmul_cuda_forward_kernel(
        Stride<scalar_t, 4> ret,         // (batch, head, time_query, feat)
        const Stride<scalar_t, 4> score, // (batch, head, time_query, restrict)
        const Stride<scalar_t, 4> value, // (batch, head, time_value, feat)
        const size_t parallel_size
        )
    {
        const auto head_size = score.sizes[1];
        const auto time_query = score.sizes[2];
        const auto restrict_size = score.sizes[3];
        const auto feat_size = value.sizes[3];
        const ptrdiff_t window = (restrict_size - 1) / 2;

        // parallel for (batch, head, time, feat)
        PARALLEL_FOR(i, parallel_size) {
            const ptrdiff_t batch_head_index = i / (time_query * feat_size);
            const ptrdiff_t remain = i % (time_query * feat_size);
            const ptrdiff_t time_query_index = remain / feat_size;
            const ptrdiff_t feat_index = remain % feat_size;
            const ptrdiff_t batch_index = batch_head_index / head_size;
            const ptrdiff_t head_index = batch_head_index % head_size;

            const auto* value_i = value.pointer({batch_index, head_index, time_query_index, feat_index});
            const auto* score_i = score.pointer({batch_index, head_index, time_query_index});

            // TODO parallel reduction
            scalar_t acc = 0;
            for (ptrdiff_t w = -window; w <= window; ++w) {
                const ptrdiff_t time_value_index = time_query_index + w;
                if (time_value_index < 0) continue;
                if (time_value_index >= value.sizes[2]) break;

                acc += score_i[(w + window) * score.strides[3]] * value_i[w * value.strides[2]];
            }
            *ret.pointer({batch_index, head_index, time_query_index, feat_index}) = acc;
        }
    }

    template <typename scalar_t>
    __global__ void unfold_matmul_cuda_backward_score_kernel(
        Stride<scalar_t, 4> dscore,      // (batch, head, time_query, restrict)
        const Stride<scalar_t, 4> dret,  // (batch, head, time_query, feat)
        const Stride<scalar_t, 4> value, // (batch, head, time_value, feat)
        const size_t parallel_size
        )
    {
        const auto head_size = dscore.sizes[1];
        const auto time_query = dscore.sizes[2];
        const auto restrict_size = dscore.sizes[3];
        const auto feat_size = value.sizes[3];
        const ptrdiff_t window = (restrict_size - 1) / 2;

        // parallel for batch, head, time_query, restrict
        PARALLEL_FOR(i, parallel_size) {
            const ptrdiff_t batch_head_index = i / (time_query * restrict_size);
            const ptrdiff_t remain = i % (time_query * restrict_size);
            const ptrdiff_t time_query_index = remain / restrict_size;
            const ptrdiff_t restrict_index = remain % restrict_size;
            const ptrdiff_t batch_index = batch_head_index / head_size;
            const ptrdiff_t head_index = batch_head_index % head_size;

            const ptrdiff_t time_value_index = time_query_index + (restrict_index - window);
            if (time_value_index < 0 || value.sizes[2] <=  time_value_index) continue;

            const auto* value_i = value.pointer({batch_index, head_index, time_value_index});
            const auto* dret_i = dret.pointer({batch_index, head_index, time_query_index});

            // TODO parallel reduction
            scalar_t acc = 0;
            for (ptrdiff_t f = 0; f < feat_size; ++f) {
                acc += dret_i[f * dret.strides[3]] * value_i[f * value.strides[3]];
            }
            *dscore.pointer({batch_index, head_index, time_query_index, restrict_index}) = acc;
        }
    }


} // namespace

at::Tensor unfold_dot_cuda_forward(
    at::Tensor query,           // (batch, head, time1, feat)
    at::Tensor key,             // (batch, head, time2, feat)
    int64_t restrict_size
    )
{
    auto batch = query.size(0);
    auto head = query.size(1);
    auto time1 = query.size(2);
    auto feat = query.size(3);
    // (batch, head, time1, restrict)
    auto ret = at::zeros({batch, head, time1, restrict_size}, query.options());
    const size_t parallel_size = batch * head * time1;

    const int threads = 1024;
    const int blocks = (parallel_size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(ret.type(), "unfold_dot_forward_cuda", ([&] {
                unfold_dot_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
                    Stride<scalar_t, 4>(ret),
                    Stride<scalar_t, 4>(query),
                    Stride<scalar_t, 4>(key),

                    parallel_size);
            }));

    return ret;
}


std::array<at::Tensor, 2> unfold_dot_cuda_backward(
    at::Tensor dret,            // (batch, head, time1, restrict)
    at::Tensor query,           // (batch, head, time1, feat)
    at::Tensor key              // (batch, head, time2, feat)
    )
{
    auto batch = query.size(0);
    auto head = query.size(1);
    auto time1 = query.size(2);
    auto feat = query.size(3);
    auto restrict_size = dret.size(3);

    auto dquery = at::empty_like(query);
    auto dkey = at::empty_like(key);
    const size_t parallel_size = batch * head * time1 * feat;
    const int threads = 1024;
    const int blocks = (parallel_size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(dret.type(), "unfold_dot_backward_cuda", ([&] {
                unfold_dot_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
                    Stride<scalar_t, 4>(dquery),
                    Stride<scalar_t, 4>(dkey),

                    Stride<scalar_t, 4>(dret),
                    Stride<scalar_t, 4>(query),
                    Stride<scalar_t, 4>(key),
                    parallel_size);
            }));

    return {dquery, dkey};
}


at::Tensor unfold_matmul_cuda_forward(
    at::Tensor score,           // (batch, head, time, restrict)
    at::Tensor value            // (batch, head, time, feat)
    )
{
    auto ret = at::empty_like(value);
    const size_t parallel_size = ret.size(0) * ret.size(1) * ret.size(2) * ret.size(3);
    const int threads = 1024;
    const int blocks = (parallel_size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(ret.type(), "unfold_matmul_forward_cuda", ([&] {
                unfold_matmul_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
                    Stride<scalar_t, 4>(ret),
                    Stride<scalar_t, 4>(score),
                    Stride<scalar_t, 4>(value),
                    parallel_size);
            }));

    return ret;
}


std::array<at::Tensor, 2> unfold_matmul_cuda_backward(
    at::Tensor dret,            // (batch, head, time, feat)
    at::Tensor score,           // (batch, head, time, restrict)
    at::Tensor value            // (batch, head, time, feat)
    )
{
    auto dscore = at::zeros_like(score);
    auto dvalue = at::zeros_like(value);
    // batch x head x time, restrict
    const size_t parallel_size = dscore.size(0) * dscore.size(1) * dscore.size(2) * dscore.size(3);
    const int threads = 1024;
    const int blocks = (parallel_size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(dscore.type(), "unfold_matmul_backward_cuda", ([&] {
                unfold_matmul_cuda_backward_score_kernel<scalar_t><<<blocks, threads>>>(
                    Stride<scalar_t, 4>(dscore),
                    Stride<scalar_t, 4>(dret),
                    Stride<scalar_t, 4>(value),
                    parallel_size);
            }));

    return {dscore, dvalue};
}
