#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <array>
#include <assert.h>
#include <iostream>

namespace {

    template <typename T, int N>
    struct Stride {
        T* data;
        // ptrdiff_t* strides;
        // ptrdiff_t* sizes;
        ptrdiff_t strides[N];
        ptrdiff_t sizes[N];

        // static const int bytes = sizeof(ptrdiff_t) * N;

        Stride(const at::Tensor& t) : data(t.data<T>()) {
            static_assert(sizeof(Stride<T, N>) < sizeof(void*) * 16,
                          "CUDA kernel launch will exeed resources");
            // hipMalloc(&strides, bytes);
            // hipMalloc(&sizes, bytes);
            // hipMemcpy(strides, t.strides().data(), bytes, hipMemcpyHostToDevice);
            // hipMemcpy(sizes, t.sizes().data(), bytes, hipMemcpyHostToDevice);

            std::copy(t.strides().begin(), t.strides().end(), strides);
            std::copy(t.sizes().begin(), t.sizes().end(), sizes);
        }

        __device__
        T* pointer(std::initializer_list<ptrdiff_t> il) {
            return const_cast<T*>(static_cast<const Stride<T, N>&>(*this).pointer(il));
        }

        __device__
        const T* pointer(std::initializer_list<ptrdiff_t> il) const {
            ptrdiff_t ret = 0;
            int n = 0;
            for (auto i : il) {
                assert(0 <= i);
                assert(i < this->sizes[n]);
                ret += i * this->strides[n];
                ++n;
            }
            return data + ret;
        }

    };

#define PARALLEL_FOR(i, n) \
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

    template <typename scalar_t>
    __global__ void unfold_dot_cuda_forward_kernel(
        Stride<scalar_t, 4> ret_tensor,         // (batch, head, time1, restrict)
        const Stride<scalar_t, 4> query_tensor, // (batch, head, time1, feat)
        const Stride<scalar_t, 4> key_tensor,   // (batch, head, time2, feat)
        size_t parallel_size
        ) {
        const auto head_size = ret_tensor.sizes[1];
        const auto time_query = query_tensor.sizes[2];
        const auto time_key = key_tensor.sizes[2];
        const auto feat_size = query_tensor.sizes[3];
        const auto restrict_size = ret_tensor.sizes[3];
        const ptrdiff_t window = (restrict_size - 1) / 2;
        PARALLEL_FOR(i, parallel_size) {
            const ptrdiff_t batch_index = i / (head_size * time_query);
            const ptrdiff_t head_index = (i % (head_size * time_query)) / time_query;
            const ptrdiff_t time_query_index = i % time_query;

            auto* ret_i = ret_tensor.pointer({batch_index, head_index, time_query_index, 0});
            const auto* query_i = query_tensor.pointer({batch_index, head_index, time_query_index, 0});
            const auto* key_i = key_tensor.pointer({batch_index, head_index, time_query_index, 0});

            for (ptrdiff_t w = -window; w <= window; ++w) {
                const ptrdiff_t time_key_index = time_query_index + w;
                if (time_key_index < 0) continue;
                if (time_key_index >= time_key) break;

                // TODO parallel reduction
                scalar_t acc = 0;
                for (ptrdiff_t f = 0; f < feat_size; ++f) {
                    acc += query_i[f * query_tensor.strides[3]] * key_i[w * key_tensor.strides[2] + f * key_tensor.strides[3]];
                }
                ret_i[(w + window) * ret_tensor.strides[3]] = acc;
            }
        }
    }


    template <typename scalar_t>
    __global__ void unfold_dot_cuda_backward_kernel(
        Stride<scalar_t, 4> dquery_tensor,      // (batch, head, time1, feat)
        Stride<scalar_t, 4> dkey_tensor,        // (batch, head, time2, feat)
        const Stride<scalar_t, 4> dret_tensor,  // (batch, head, time1, restrict)
        const Stride<scalar_t, 4> query_tensor, // (batch, head, time1, feat)
        const Stride<scalar_t, 4> key_tensor,   // (batch, head, time2, feat)
        size_t parallel_size
        ) {
        const auto head_size = dret_tensor.sizes[1];
        const auto time_query = query_tensor.sizes[2];
        const auto time_key = key_tensor.sizes[2];
        const auto feat_size = query_tensor.sizes[3];
        const auto restrict_size = dret_tensor.sizes[3];

        const ptrdiff_t window = (restrict_size - 1) / 2;
        const ptrdiff_t rev_offset = restrict_size - 1 - window;

        // parallel for each (batch, head, time1, feat). sequential for each (restrict)
        PARALLEL_FOR(i, parallel_size) {
            const ptrdiff_t batch_head_index = i / (time_query * feat_size);
            const ptrdiff_t remain = i % (time_query * feat_size);
            const ptrdiff_t time_query_index = remain / feat_size;
            const ptrdiff_t feat_index = remain % feat_size;
            const ptrdiff_t batch_index = batch_head_index / head_size;
            const ptrdiff_t head_index = batch_head_index % head_size;

            const auto* dret_i = dret_tensor.pointer({batch_index, head_index, time_query_index, 0});
            const auto* query_i = query_tensor.pointer({batch_index, head_index, time_query_index, feat_index});
            const auto* key_i = key_tensor.pointer({batch_index, head_index, time_query_index, feat_index});
            scalar_t dquery_i = 0;
            scalar_t dkey_i = 0;

            // for (auto w = -min(time_query_index, window); w <= min(time_query - time_query_index + 1, window); ++w) {
            for (auto w = -window; w <= window; ++w) {
                auto t = time_query_index + w;
                if (0 <= t && t < time_key) {
                    dquery_i += dret_i[(w + window) * dret_tensor.strides[3]] * key_i[w * key_tensor.strides[2]];
                    dkey_i += dret_i[w * dret_tensor.strides[2] + (rev_offset - w) * dret_tensor.strides[3]]
                        * query_i[w * query_tensor.strides[2]];
                }
            }
            *dquery_tensor.pointer({batch_index, head_index, time_query_index, feat_index}) = dquery_i;
            *dkey_tensor.pointer({batch_index, head_index, time_query_index, feat_index}) = dkey_i;
        }
    }

} // namespace

at::Tensor unfold_dot_cuda_forward(
    at::Tensor query,           // (batch, head, time1, feat)
    at::Tensor key,             // (batch, head, time2, feat)
    int64_t restrict_size
    )
{
    auto batch = query.size(0);
    auto head = query.size(1);
    auto time1 = query.size(2);
    auto feat = query.size(3);
    // (batch, head, time1, restrict)
    auto ret = at::zeros({batch, head, time1, restrict_size}, query.options());
    const size_t parallel_size = batch * head * time1;

    const int threads = 1024;
    const int blocks = (parallel_size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(ret.type(), "unfold_dot_forward_cuda", ([&] {
                unfold_dot_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
                    Stride<scalar_t, 4>(ret),
                    Stride<scalar_t, 4>(query),
                    Stride<scalar_t, 4>(key),

                    parallel_size);
            }));

    return ret;
}


std::array<at::Tensor, 2> unfold_dot_cuda_backward(
    at::Tensor dret,            // (batch, head, time1, restrict)
    at::Tensor query,           // (batch, head, time1, feat)
    at::Tensor key              // (batch, head, time2, feat)
    )
{
    auto batch = query.size(0);
    auto head = query.size(1);
    auto time1 = query.size(2);
    auto feat = query.size(3);
    auto restrict_size = dret.size(3);

    auto dquery = at::empty_like(query);
    auto dkey = at::empty_like(key);
    const size_t parallel_size = batch * head * time1 * feat;
    const int threads = 1024;
    const int blocks = (parallel_size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(dret.type(), "unfold_dot_backward_cuda", ([&] {
                unfold_dot_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
                    Stride<scalar_t, 4>(dquery),
                    Stride<scalar_t, 4>(dkey),

                    Stride<scalar_t, 4>(dret),
                    Stride<scalar_t, 4>(query),
                    Stride<scalar_t, 4>(key),
                    parallel_size);
            }));

    return {dquery, dkey};
}

